// This example is taken from https://cuda-tutorial.readthedocs.io/en/latest/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello(){
  printf("Hello World from GPU!\n");
}

int main() {

  printf("Hello World from CPU!\n");

  cuda_hello<<<1,1>>>();

  hipDeviceSynchronize();
  return 0;
}
