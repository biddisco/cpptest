// This example is taken from https://cuda-tutorial.readthedocs.io/en/latest/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello(){
  int d;
  hipGetDevice(&d);
  printf("Hello World from GPU device %d\n",d);
}

int main() {

  printf("Hello World from CPU!\n");
  

  cuda_hello<<<1,1>>>();

  hipDeviceSynchronize();
  return 0;
}
